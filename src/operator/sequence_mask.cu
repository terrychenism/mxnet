#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file sequence_mask.cu
 * \brief
 * \author Sebastian Bodenstein
*/

#include "./sequence_mask-inl.h"


namespace mshadow {
namespace cuda {

////////////////////////////////////////////////////////////////////////////////
// Cross-Entropy loss
template<int n_bits, typename DType>
__global__ void SequenceMaskKernel(Tensor<gpu, 3, DType> dst,
                    const Tensor<gpu, 1, DType> lengths, DType value) {
  const index_t smax = dst.size(0);
  const index_t bmax = lengths.size(1);
  const index_t nmax = dst.size(2);
  unsigned int batch = threadIdx.x + blockIdx.x * blockDim.x;

  // early return if out of bounds
  if (batch >= bmax)
    return;

  // loop over batches
    for (index_t s = lengths[batch]; s < smax; ++s)
      for (index_t r = 0; r < nmax; ++r)
      dst[s][batch][r] = value;
}

////////////////////////////////////////////////////////////////////////////////

template<typename DType>
inline void SequenceMask(const Tensor<gpu, 3, DType> &dst,
                         const Tensor<gpu, 1, DType> &lengths, DType value) {
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(dst.size(1));
  CheckLaunchParam(dimGrid, dimBlock, "SequenceMask");
  hipStream_t stream = Stream<gpu>::GetStream(dst.stream_);
  SequenceMaskKernel<kBaseThreadBits, DType><<<dimGrid, dimBlock, 0, stream>>>(dst, lengths, value);
}

////////////////////////////////////////////////////////////////////////////////
}  // namespace cuda

template<typename DType>
inline void SequenceMask(Tensor<gpu, 3, DType> dst,
                   const Tensor<gpu, 1, DType> &lengths, DType value) {
  cuda::SequenceMask(dst, lengths, value);
}

}  // namespace mshadow

////////////////////////////////////////////////////////////////////////////////

namespace mxnet {
namespace op {
template <> Operator *CreateOp<gpu>(SequenceMaskParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType,
                           { op = new SequenceMaskOp<gpu, DType>(param); })
  return op;
}

}  // namespace op
}  // namespace mxnet
